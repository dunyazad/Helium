#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <Helium/Graphics/Geometry/Geometry.cuh>

#include <Helium/Graphics/Geometry/HeGeometryTriangleSoup.h>
#include <Helium/Graphics/HeVertexBufferObject.hpp>
#include <Helium/Core/Project/HeCameraInfo.h>

namespace ArtificialNature {
	
	union Triangle {
		struct {
			glm::vec3 v0;
			glm::vec3 v1;
			glm::vec3 v2;
		};
		struct {
			glm::vec3 n0;
			glm::vec3 n1;
			glm::vec3 n2;
		};
	};

	__global__ void CUDA_ComputeFaceNormals(Triangle* inputFaces, Triangle* outputFaceNormals, int faceCount)
	{
		int threadID = blockDim.x * blockIdx.x + threadIdx.x;

		if (threadID < faceCount)
		{
			//glm::vec3 d01 = glm::normalize(inputFaces[threadID].v1 - inputFaces[threadID].v0);
			//glm::vec3 d21 = glm::normalize(inputFaces[threadID].v2 - inputFaces[threadID].v0);
			//glm::vec3 n = glm::cross(d01, d21);

			//outputFaceNormals[threadID].n0 = n;
			//outputFaceNormals[threadID].n1 = n;
			//outputFaceNormals[threadID].n2 = n;

			outputFaceNormals[threadID].n0 = inputFaces[threadID].v0;
			outputFaceNormals[threadID].n1 = inputFaces[threadID].v1;
			outputFaceNormals[threadID].n2 = inputFaces[threadID].v2;
		}
	}

	void ComputeFaceNormals(HeGeometryTriangleSoup* pGeometry)
	{
		int faceCount = pGeometry->GetFaceCount();

		auto vbo = pGeometry->GetVBO();
		auto elements = vbo->GetElements();


		Triangle* inputFaces;
		Triangle* outputFaceNormals;

		hipMalloc((void**)&inputFaces, faceCount * sizeof(Triangle));
		hipMalloc((void**)&outputFaceNormals, faceCount * sizeof(Triangle));

		hipMemcpy(inputFaces, &(elements[0]), faceCount * sizeof(Triangle), hipMemcpyHostToDevice);

		dim3 blockSize(512, 1, 1);
		dim3 gridSize(512 / faceCount + 1, 1, 1);

		CUDA_ComputeFaceNormals << <gridSize, blockSize >> > (inputFaces, outputFaceNormals, faceCount);

		vector<glm::vec3> vertexNormals;
		vertexNormals.resize(faceCount * 3);

		hipMemcpy(&vertexNormals[0], outputFaceNormals, faceCount * sizeof(Triangle), hipMemcpyDeviceToHost);

		auto vnbo = pGeometry->GetVNBO();
		vnbo->SetElements(vertexNormals);
	}

	__global__ void CUDA_RayIntersect(Triangle* inputFaces, bool* outputIntersection, int faceCount, const glm::vec3& rayOrigin, const glm::vec3& rayDirection)
	{
		int threadID = blockDim.x * blockIdx.x + threadIdx.x;

		if (threadID < faceCount)
		{
			glm::vec2 baricenter;
			float distance = 0.0f;
			if (glm::intersectRayTriangle(rayOrigin, rayDirection, inputFaces[threadID].v0, inputFaces[threadID].v1, inputFaces[threadID].v2, baricenter, distance))
			{
				if (distance > 0) {
					outputIntersection[threadID] = true;
				}
			}
		}
	}

	void RayIntersect(HeGeometryTriangleSoup* pGeometry, const glm::vec3& rayOrigin, const glm::vec3& rayDirection)
	{
		int faceCount = pGeometry->GetFaceCount();

		auto vbo = pGeometry->GetVBO();
		auto elements = vbo->GetElements();

		Triangle* inputFaces;
		bool* outputIntersection;

		hipMalloc((void**)&inputFaces, faceCount * sizeof(Triangle));
		hipMalloc((void**)&outputIntersection, faceCount * sizeof(bool));

		hipMemcpy(inputFaces, &(elements[0]), faceCount * sizeof(Triangle), hipMemcpyHostToDevice);

		dim3 blockSize(512, 1, 1);
		dim3 gridSize(512 / faceCount + 1, 1, 1);

		CUDA_RayIntersect << <gridSize, blockSize >> > (inputFaces, outputIntersection, faceCount, rayOrigin, rayDirection);

		vector<bool> results;
		results.resize(faceCount);

		hipMemcpy(&results[0], outputIntersection, faceCount * sizeof(bool), hipMemcpyDeviceToHost);
	}

	__global__ void CUDA_ComputeOcclusionInfo(Triangle* inputFaces, bool* outputIntersection, int faceCount, glm::vec3* camInfos, int camInfoCount)
	{
		int threadID = blockDim.x * blockIdx.x + threadIdx.x;

		if (threadID < faceCount)
		{
			for (size_t j = 0; j < camInfoCount / 2; j++)
			{
				auto cameraPosition = camInfos[j * 2];
				auto cameraFront = camInfos[j * 2 + 1];

				for (int i = 0; i < faceCount; i++)
				{
					glm::vec2 baricenter;
					float distance = 0.0f;
					if (glm::intersectRayTriangle(cameraPosition, cameraFront, inputFaces[i].v0, inputFaces[i].v1, inputFaces[i].v2, baricenter, distance))
					{
						if (distance > 0) {
							outputIntersection[threadID] = true;
						}
					}
				}
			}
		}
	}

	void ComputeOcclusionInfo(HeGeometryTriangleSoup* pGeometry, const vector<const HeCameraInfo*>& cameraInfos)
	{
		int faceCount = pGeometry->GetFaceCount();

		auto vbo = pGeometry->GetVBO();
		auto elements = vbo->GetElements();

		Triangle* inputFaces_dev;

		hipMalloc((void**)&inputFaces_dev, faceCount * sizeof(Triangle));
		hipMemcpy(inputFaces_dev, &(elements[0]), faceCount * sizeof(Triangle), hipMemcpyHostToDevice);

		bool* outputIntersection_dev;
		hipMalloc((void**)&outputIntersection_dev, faceCount * cameraInfos.size() * sizeof(bool));

		vector<glm::vec3> cam_infos;
		cam_infos.resize(cameraInfos.size() * 2);
		for (size_t i = 0; i < cameraInfos.size(); i++)
		{
			auto info = cameraInfos[i];
			auto cameraPosition = info->GetPosition();
			auto cameraFront = glm::vec3(info->GetViewMatrix()[3]);
			cam_infos[i * 2] = cameraPosition;
			cam_infos[i * 2 + 1] = cameraFront;
		}

		glm::vec3* cameraInfos_dev;
		hipMalloc((void**)&cameraInfos_dev, cameraInfos.size() * sizeof(glm::vec3) * 2);
		hipMemcpy(cameraInfos_dev, &(cameraInfos_dev[0]), cameraInfos.size() * sizeof(glm::vec3) * 2, hipMemcpyHostToDevice);


		dim3 blockSize(512, 1, 1);
		dim3 gridSize(512 / faceCount + 1, 1, 1);

		CUDA_ComputeOcclusionInfo << <gridSize, blockSize >> > (inputFaces_dev, outputIntersection_dev, faceCount, cameraInfos_dev, cameraInfos.size() * 2);

		vector<bool> results;
		results.resize(faceCount * cameraInfos.size());
		hipMemcpy(&results[0], outputIntersection_dev, faceCount * cameraInfos.size() * sizeof(bool), hipMemcpyDeviceToHost);
	}
}
